
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10 // Size of the vectors

__global__ void vectorAdd(int *a, int *b, int *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
        c[index] = a[index] + b[index];
}

int main() {
    int *a, *b, *c; // Host vectors
    int *d_a, *d_b, *d_c; // Device vectors

    // Allocate memory on the host
    a = (int *)malloc(N * sizeof(int));
    b = (int *)malloc(N * sizeof(int));
    c = (int *)malloc(N * sizeof(int));

    // Allocate memory on the device
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));

    // Initialize vectors on the host
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Copy vectors from host to device
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Perform vector addition
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Output some elements of the resulting vector
    printf("First 10 elements of the resulting vector:\n");
    for (int i = 0; i < 10; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
